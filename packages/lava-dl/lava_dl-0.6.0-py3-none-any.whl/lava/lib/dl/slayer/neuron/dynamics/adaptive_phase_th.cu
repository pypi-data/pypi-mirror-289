#include "hip/hip_runtime.h"
/*
Copyright (C) 2021 Intel Corporation
SPDX-License-Identifier:  BSD-3-Clause
*/

#include <torch/script.h>
#include <torch/all.h>
#include <torch/extension.h>

#include <iostream>
#include <memory>
#include <vector>

#define CHECK_CUDA(x) AT_ASSERTM(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CHECK_DEVICE(x, y) AT_ASSERTM(x.device().index() == y.device().index(), #x " and " #y " must be in same CUDA device")

using namespace torch::indexing;
using torch::autograd::AutogradContext;
using torch::autograd::Variable;
using torch::autograd::variable_list;

template <class T>
__global__ void APThDynamicsFwdKernel(
    T* __restrict__ threshold,
    T* __restrict__ refractory,
    const T* __restrict__ re_input,
    const T* __restrict__ im_input,
    const T* __restrict__ im_state,
    const T* __restrict__ ref_state,
    const T* __restrict__ ref_decay,
    const T* __restrict__ th_state,
    const T* __restrict__ th_decay,
    const int th_scale,
    const int th0,
    const int w_scale,
    const int num_neurons,
    const int neurons_per_batch,
    const int num_decays, // this determines individual, channelwise or shared decay
    const int decay_block, // if decay_block==1 then its individual neuron
    const int num_steps
) {
    unsigned neuron_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(neuron_id >= num_neurons)    return;

    int th = th_state[neuron_id] * w_scale;
    int th_step = th_scale;
    int th_decay_int;
    int ref_new = ref_state[neuron_id] * w_scale;
    int ref_decay_int;
    int linear_id; 
    float imag_input_old = im_state[neuron_id];

    if(num_decays > 1) {  
        // individual decays or channelwise decay
        // num_decays * decay_block == num_neurons
        th_decay_int = (1<<12) - th_decay[(neuron_id) % neurons_per_batch / decay_block];
        ref_decay_int = (1<<12) - ref_decay[(neuron_id) % neurons_per_batch / decay_block];
    } else { // shared decays
        th_decay_int = (1<<12) - th_decay[0];
        ref_decay_int = (1<<12) - ref_decay[0];
    }

    // if(neuron_id == 0)  printf("int: %d bytes\n", sizeof(int));

    for(int n=0; n<num_steps; ++n) {
        linear_id = n + neuron_id * num_steps;

        ref_new = (ref_new * ref_decay_int) >> 12;
        th = (((th - th0) * th_decay_int) >> 12) + th0;
        
        threshold[linear_id] = 1.0f * th / w_scale;
        refractory[linear_id] = 1.0f * ref_new / w_scale;
        
        if(
            int(w_scale * re_input[linear_id]) >= (th + ref_new) &&
            imag_input_old < 0 &&
            im_input[linear_id] >= 0
        ) {
            ref_new += 2*th;
            th += th_step;
        } 
        imag_input_old = im_input[linear_id];
    }
}

variable_list APThDynamicsFwd(
    const Variable re_input,
    const Variable im_input,
    const Variable im_state,
    const Variable ref_state,
    const Variable ref_decay,
    const Variable th_state,
    const Variable th_decay,
    float th_scale,
    float th0,
    int w_scale
) {
    // make sure all the inputs are contigious and in same device
    CHECK_INPUT(re_input);
    CHECK_INPUT(im_input);
    CHECK_INPUT(im_state);
    CHECK_INPUT(th_state);
    CHECK_INPUT(th_decay);
    CHECK_INPUT(ref_state);
    CHECK_INPUT(ref_decay);
    CHECK_DEVICE(re_input, im_input);
    CHECK_DEVICE(re_input, im_state);
    CHECK_DEVICE(re_input, th_state);
    CHECK_DEVICE(re_input, th_decay);
    CHECK_DEVICE(re_input, ref_state);
    CHECK_DEVICE(re_input, ref_decay);
    hipSetDevice(re_input.device().index());

    auto device = re_input.device().type();
    auto dtype = re_input.dtype();
    auto threshold = torch::zeros_like(re_input);
    auto refractory = torch::zeros_like(re_input);

    int num_neurons = re_input.numel() / re_input.size(-1);
    int thread = 256;
    int block = ceil(1.0f * num_neurons / thread);

    // std::cout << "num_neurons : " << num_neurons << std::endl
    //           << "thread : " << thread << std::endl
    //           << "block : " << block << std::endl;
    APThDynamicsFwdKernel<float><<< block, thread >>>(
        threshold.data<float>(),
        refractory.data<float>(),
        re_input.data<float>(), 
        im_input.data<float>(), 
        im_state.data<float>(), 
        ref_state.data<float>(),
        ref_decay.data<float>(),
        th_state.data<float>(),
        th_decay.data<float>(),
        th_scale*w_scale, th0*w_scale, w_scale, num_neurons, num_neurons / re_input.size(0),
        th_decay.numel(), // num_th_decays 
        num_neurons / th_decay.numel() / re_input.size(0), // decay_block 
        re_input.size(-1) // num_steps
    );
    // hipDeviceSynchronize();

    return {threshold, refractory};
}

class APThDynamics : public torch::autograd::Function<APThDynamics> {
    public:
    static variable_list forward(
        AutogradContext* ctx, 
        const Variable re_input,
        const Variable im_input,
        const Variable im_state,
        const Variable ref_state,
        const Variable ref_decay,
        const Variable th_state,
        const Variable th_decay,
        float th_scale,
        float th0,
        int w_scale
    ) {
        auto result = APThDynamicsFwd(re_input, im_input, im_state, ref_state, ref_decay, th_state, th_decay, th_scale, th0, w_scale);
        return result;
    }

    static variable_list backward(AutogradContext* ctx, variable_list grad_output) {
        // No backward gradients from here
        return {
            torch::Tensor(), // Gradients of non-tensor arguments to forward must be `torch::Tensor()`.
            torch::Tensor(), 
            torch::Tensor(), 
            torch::Tensor(), 
            torch::Tensor(), 
            torch::Tensor(), 
            torch::Tensor(), 
            torch::Tensor(), 
            torch::Tensor(), 
            torch::Tensor() 
        };
    }
};

std::vector<torch::Tensor> APThDynamicsFx(
    const torch::Tensor& re_input,
    const torch::Tensor& im_input,
    const torch::Tensor& im_state,
    const torch::Tensor& ref_state,
    const torch::Tensor& ref_decay,
    const torch::Tensor& th_state,
    const torch::Tensor& th_decay,
    float th_scale,
    float th0,
    int w_scale
) {
    auto result = APThDynamics::apply(
            re_input, im_input, im_state,
            ref_state, ref_decay, 
            th_state, th_decay, th_scale, th0, 
            w_scale
        );
    return {result[0], result[1]};
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
	m.def("dynamics", &APThDynamicsFx, "Dynamics of Loihi Adaptive Phase Threshold.");
    m.def("fwd", &APThDynamicsFwd, "Fwd dynamics of Loihi Adaptive Phase Threshold.");
    // m.def("bwd", &APThDynamicsBwd, "Bwd dynamics of Loihi Adaptive Phase Threshold.");
}