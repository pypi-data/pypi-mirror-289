#include "hip/hip_runtime.h"
// MIT License

// Copyright (c) 2024 dechin

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

// nvcc -shared ./FES.cu -Xcompiler -fPIC -o ./libcufes.so
#include "./FES.cuh"

constexpr int THREADS = 64;
constexpr double kT = 8.314 * 300 / 4184;

__global__ void WeightKernel(int CV_LENGTH, double* bias, double* V, double shift, double* weight){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  
    if (idx < CV_LENGTH){
        V[idx] = bias[idx] / kT - shift;
        weight[idx] = expf(V[idx]);
    }
}

extern "C" int GetWeight(int CV_LENGTH, double* bias, double shift, double* weight){
    double *bias_device, *V_device, *weight_device;
    hipMalloc(&bias_device, CV_LENGTH * sizeof(double));  
    hipMalloc(&V_device, CV_LENGTH * sizeof(double));  
    hipMalloc(&weight_device, CV_LENGTH * sizeof(double));  
    hipMemcpy(bias_device, bias, CV_LENGTH * sizeof(double), hipMemcpyHostToDevice); 
    int numBlocks = CV_LENGTH / THREADS + 1;
    WeightKernel<<<numBlocks, THREADS>>>(CV_LENGTH, bias_device, V_device, shift, weight_device); 
    hipDeviceSynchronize();
    hipMemcpy(weight, weight_device, CV_LENGTH * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(bias_device);  
    hipFree(V_device);
    hipFree(weight_device); 
    hipDeviceReset();
    return 1;
}

__global__ void DistKernel(int CV_LENGTH, CRD* crd, PATH* cv, double* dis){
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    double tmp = 0.0;
    if (idx < CV_LENGTH){
        tmp += pow(crd[0].x - cv[idx].crds.x, 2);
        tmp += pow(crd[0].y - cv[idx].crds.y, 2);
        tmp += pow(crd[0].z - cv[idx].crds.z, 2);
        dis[idx] = sqrtf(tmp);
    }
}

extern "C" int GetDist(int CV_LENGTH, CRD* crd, PATH* cv, double* dis){
    CRD* crd_device;
    PATH* cv_device;
    double *dis_device;
    hipMalloc(&crd_device, sizeof(CRD));
    hipMalloc(&cv_device, CV_LENGTH*sizeof(PATH));
    hipMalloc(&dis_device, CV_LENGTH*sizeof(double));
    hipMemcpy(crd_device, crd, sizeof(CRD), hipMemcpyHostToDevice);
    hipMemcpy(cv_device, cv, CV_LENGTH*sizeof(PATH), hipMemcpyHostToDevice);
    int numBlocks = CV_LENGTH / THREADS + 1;
    DistKernel<<<numBlocks, THREADS>>>(CV_LENGTH, crd_device, cv_device, dis_device);
    hipDeviceSynchronize();
    hipMemcpy(dis, dis_device, CV_LENGTH * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(crd_device);
    hipFree(cv_device);
    hipFree(dis_device);
    hipDeviceReset();
    return 1;
}

__global__ void GaussDistKernel(int CV_LENGTH, CRD* crd, PATH* cv, double* dis){
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    double tmp = 0.0;
    if (idx < CV_LENGTH){
        tmp -= pow(crd[0].x - cv[idx].crds.x, 2);
        tmp -= pow(crd[0].y - cv[idx].crds.y, 2);
        tmp -= pow(crd[0].z - cv[idx].crds.z, 2);
        tmp /= 2.0;
        dis[idx] = expf(tmp);
    }
}

extern "C" int GaussGetDist(int CV_LENGTH, CRD* crd, PATH* cv, double* dis){
    CRD* crd_device;
    PATH* cv_device;
    double *dis_device;
    hipMalloc(&crd_device, sizeof(CRD));
    hipMalloc(&cv_device, CV_LENGTH*sizeof(PATH));
    hipMalloc(&dis_device, CV_LENGTH*sizeof(double));
    hipMemcpy(crd_device, crd, sizeof(CRD), hipMemcpyHostToDevice);
    hipMemcpy(cv_device, cv, CV_LENGTH*sizeof(PATH), hipMemcpyHostToDevice);
    int numBlocks = CV_LENGTH / THREADS + 1;
    GaussDistKernel<<<numBlocks, THREADS>>>(CV_LENGTH, crd_device, cv_device, dis_device);
    hipDeviceSynchronize();
    hipMemcpy(dis, dis_device, CV_LENGTH * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(crd_device);
    hipFree(cv_device);
    hipFree(dis_device);
    hipDeviceReset();
    return 1;
}

__global__ void GaussDistHeightKernel(int CV_LENGTH, CRD* crd, PATH* cv, double* dis, double* height){
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    double tmp = 0.0;
    if (idx < CV_LENGTH){
        tmp -= pow(crd[0].x - cv[idx].crds.x, 2);
        tmp -= pow(crd[0].y - cv[idx].crds.y, 2);
        tmp -= pow(crd[0].z - cv[idx].crds.z, 2);
        tmp /= 2.0;
        dis[idx] = expf(tmp) * height[idx];
    }
}

extern "C" int GaussGetDistHeight(int CV_LENGTH, CRD* crd, PATH* cv, double* dis, double* height){
    CRD* crd_device;
    PATH* cv_device;
    double *dis_device;
    double *height_device;
    hipMalloc(&crd_device, sizeof(CRD));
    hipMalloc(&cv_device, CV_LENGTH*sizeof(PATH));
    hipMalloc(&dis_device, CV_LENGTH*sizeof(double));
    hipMalloc(&height_device, CV_LENGTH*sizeof(double));
    hipMemcpy(crd_device, crd, sizeof(CRD), hipMemcpyHostToDevice);
    hipMemcpy(height_device, height, CV_LENGTH*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cv_device, cv, CV_LENGTH*sizeof(PATH), hipMemcpyHostToDevice);
    int numBlocks = CV_LENGTH / THREADS + 1;
    GaussDistHeightKernel<<<numBlocks, THREADS>>>(CV_LENGTH, crd_device, cv_device, dis_device, height_device);
    hipDeviceSynchronize();
    hipMemcpy(dis, dis_device, CV_LENGTH * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(crd_device);
    hipFree(cv_device);
    hipFree(dis_device);
    hipFree(height_device);
    hipDeviceReset();
    return 1;
}
